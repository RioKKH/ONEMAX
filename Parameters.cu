#include <iostream>
#include <cstdio>
#include <fstream>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Parameters.h"
#include "GAregex.hpp"


void Parameters::loadParams()
{
    std::ifstream infile(PARAMNAME);
    std::string line;
    std::smatch results;

    while (getline(infile, line))
    {
        if (std::regex_match(line, results, rePOPSIZE))
        {
            cpuEvoPrms.POPSIZE = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reCHROMOSOME))
        {
            cpuEvoPrms.CHROMOSOME_ACTUAL = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reNUM_OF_GENERATIONS))
        {
            cpuEvoPrms.NUM_OF_GENERATIONS = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reNUM_OF_ELITE))
        {
            cpuEvoPrms.NUM_OF_ELITE = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reTOURNAMENT_SIZE))
        {
            cpuEvoPrms.TOURNAMENT_SIZE = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reNUM_OF_CROSSOVER_POINTS))
        {
            cpuEvoPrms.NUM_OF_CROSSOVER_POINTS = std::stoi(results[1].str());
        }
        else if (std::regex_match(line, results, reMUTATION_RATE))
        {
            cpuEvoPrms.MUTATION_RATE = std::stof(results[1].str());
        }
    }

    //- ここで確保する遺伝子メモリサイズを決定する
    for (int i = 5; i <= 10; ++i)
    {
        if (cpuEvoPrms.CHROMOSOME_ACTUAL == (1 << i))
        {
            cpuEvoPrms.CHROMOSOME_PSEUDO = (1 << i);
        }
        else if (( (1 << i) < cpuEvoPrms.CHROMOSOME_ACTUAL )
                && ( cpuEvoPrms.CHROMOSOME_ACTUAL < (1 << (i + 1)) ))
        {
            cpuEvoPrms.CHROMOSOME_PSEUDO = (1 << (i + 1));
        }
    }

    //-  総遺伝子長と総遺伝子サイズを設定する
    cpuEvoPrms.N_ACTUAL = cpuEvoPrms.POPSIZE * cpuEvoPrms.CHROMOSOME_ACTUAL;
    cpuEvoPrms.N_PSEUDO = cpuEvoPrms.POPSIZE * cpuEvoPrms.CHROMOSOME_PSEUDO;
    cpuEvoPrms.Nbytes_ACTUAL = cpuEvoPrms.N_ACTUAL * sizeof(int);
    cpuEvoPrms.Nbytes_PSEUDO = cpuEvoPrms.N_PSEUDO * sizeof(int);

    infile.close();

    return;
}

int Parameters::getPopsize() const { return cpuEvoPrms.POPSIZE; }
int Parameters::getChromosomeActual() const { return cpuEvoPrms.CHROMOSOME_ACTUAL; }
int Parameters::getChromosomePseudo() const { return cpuEvoPrms.CHROMOSOME_PSEUDO; }
int Parameters::getNumOfGenerations() const { return cpuEvoPrms.NUM_OF_GENERATIONS; }
int Parameters::getNumOfElite() const { return cpuEvoPrms.NUM_OF_ELITE; }
int Parameters::getTournamentSize() const { return cpuEvoPrms.TOURNAMENT_SIZE; }
int Parameters::getNumOfCrossoverPoints() const { return cpuEvoPrms.NUM_OF_CROSSOVER_POINTS; }
float Parameters::getMutationRate() const { return cpuEvoPrms.MUTATION_RATE; }
int Parameters::getNActual() const { return cpuEvoPrms.N_ACTUAL; }
int Parameters::getNPseudo() const { return cpuEvoPrms.N_PSEUDO; }
int Parameters::getNbytesActual() const { return cpuEvoPrms.Nbytes_ACTUAL; }
int Parameters::getNbytesPseudo() const { return cpuEvoPrms.Nbytes_PSEUDO; }
EvolutionParameters Parameters::getEvoPrms() const { return cpuEvoPrms; }

void Parameters::showParams() const
{
    std::cout << "POPSIZE: " << cpuEvoPrms.POPSIZE << std::endl;
    std::cout << "CHROMOSOME: " << cpuEvoPrms.CHROMOSOME_ACTUAL << std::endl;
    std::cout << "NUM_OF_GENERATIONS: " << cpuEvoPrms.NUM_OF_GENERATIONS << std::endl;
    std::cout << "NUM_OF_ELITE: " << cpuEvoPrms.NUM_OF_ELITE << std::endl;
    std::cout << "TOURNAMENT_SIZE: " << cpuEvoPrms.TOURNAMENT_SIZE << std::endl;
    std::cout << "NUM_OF_CROSSOVER_POINTS: " << cpuEvoPrms.NUM_OF_CROSSOVER_POINTS << std::endl;
    std::cout << "MUTATION_RATE: " << cpuEvoPrms.MUTATION_RATE << std::endl;
    std::cout << "N: " << cpuEvoPrms.N_ACTUAL << std::endl;
    std::cout << "Nbytes: " << cpuEvoPrms.Nbytes_ACTUAL << std::endl;
}

